#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
#define ROW 10
#define COLUMN 11

#define ARRAY_BYTES (ROW * COLUMN * sizeof(float))
#define NUMTHREADS 10
#define NUMBLOCKS 1

__global__ void gauss(float** d_out,float** d_in)
{
	int x = threadIdx.x;
	for (int i = x; i < 10; i++)
	{
		float factor = d_in[i][x]/d_in[x][x];
		//printf("%f factor\n", factor);
		for(int p=x;p<11;p++)
        {		
		d_out[i][p] = d_in[i][p] - (factor * (d_in[x][p]));
              //printf("%f output\n",  d_out[i][p]);
		
		}
	}



}
void printans(float output[][COLUMN])
{
for (int o = 0; o < 10; o++)
		{for (int i = 0; i < 11; i++)
			printf("%f " , output[o][i]);
			printf("\n");}
				
}

int main(int argc, char** argv)
{ 
	float random[10][11];
	float** d_in;
	float** d_out;
	float ranoutput[10][11];
	// generating matrix
	for (int o = 0; o < 10; o++)
		for (int i = 0; i < 11; i++)
		 random[o][1] = rand()%100;
	
	//allocating memory

	hipMalloc((void**)&d_in, ARRAY_BYTES);
	hipMalloc((void**)&d_out, ARRAY_BYTES);
	hipMemcpy(d_in, random, ARRAY_BYTES, hipMemcpyHostToDevice);

   //launching kernel

	gauss << <NUMBLOCKS, NUMTHREADS >> > (d_out,d_in);
	hipDeviceSynchronize();
	hipMemcpy(d_out, ranoutput, ARRAY_BYTES, hipMemcpyDeviceToHost);
	
	// print output
	printans(ranoutput);

    hipFree(d_in);
	hipFree(d_out);
	return 0;

}
